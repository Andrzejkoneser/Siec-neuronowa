#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c)
{
    //obliczam numer w�tku na podstawie numeru bloku, w�tku w bloku i ilo�ci w�tk�w w bloku
    
    //tych "w�tk�w" nie rozumiem, rozumiem podawanie rozmiaru bloku, ale takie kody s� w internecie i dzia�aj�
    //wi�c te� takich u�ywam
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

#define N 100

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	//alokuj� pami�� w karcie graficznej

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	//alokuj� pamie� w procesorze

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = i;
		b[i] = i*5
		c[i] = 0;
	}

	//kopiuj� zawarto�� z procesorka do karty graicznej
	
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	//podaj� rozmiar bloku na karcie //block_size - rozmiar bloku
	
	int block_size = 4;
	
	// Obliczenie liczby blok�w koniczecznych do przetworzenia ca�ej tablicy  //n_blocks - liczba blok�w
	
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);  /*jak nieprawda to dodam jeszcze jeden blok,
	jak prawda to nie dodam, bo mie�ci si� w ilo�ci blok�w przy dzieleniu (bez reszty) *\
	
	add<<< (n_blocks, block_size>>>( d_a, d_b, d_c );
	//add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	//kopiuj� warto�ci z tablicy c z karty graficznej do c na procesorze
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	//wyniki mojej pracy, sprawdzenie czy si� uda�o
	int i;
	for (i=0; i<N; i++) {
	printf( "c[%i] = %d\n",i,c[i] );
	}

	//zwalniam pamie� w karcie i na procesorze
	
	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
}
